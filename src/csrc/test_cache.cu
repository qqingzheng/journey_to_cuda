#include <hip/hip_runtime.h>
#include <iostream>

#define N (4294967296 / 4)

__global__ void cache_test(int *data, int stride)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int accessIdx = ((long long)idx * stride) % N;
    if (accessIdx < N)
    {
        data[accessIdx] += 1;
    }
}

int main()
{
    hipError_t cudaStatus;
    int *d_data;
    cudaStatus = hipMalloc(&d_data, N * sizeof(int));
    printf("Memory allocated: %f MB\n", (float)N * sizeof(int) / 1024 / 1024);
    if (cudaStatus != hipSuccess)
    {
        printf("hipMalloc failed, %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }
    hipMemset(d_data, 0, N * sizeof(int));
    for (int stride = 1; stride <= 1024 * 1024 / sizeof(int); stride *= 10)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        int blockSize = 32;
        int gridSize = (N + blockSize - 1) / blockSize;
        printf("stride: %d, gridSize: %d, blockSize: %d\n", stride, gridSize, blockSize);
        cache_test<<<gridSize, blockSize>>>(d_data, stride);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
            return 1;
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time: %.3f ms\n", milliseconds);
    }
    hipFree(d_data);
    return 0;
}