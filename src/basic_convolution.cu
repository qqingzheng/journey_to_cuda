#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>

#define INPUT_CHANNEL 3
#define INPUT_HEIGHT 10240
#define INPUT_WIDTH 10240
#define KERNEL_NUMBER 16
#define KERNEL_CHANNEL 3
#define KERNEL_HEIGHT 3
#define KERNEL_WIDTH 3
#define OUTPUT_HEIGHT (INPUT_HEIGHT - KERNEL_HEIGHT + 1)
#define OUTPUT_WIDTH (INPUT_WIDTH - KERNEL_WIDTH + 1)

__global__ void convolution(
    double *input,
    double *output,
    double *kernel,
    int inputHeight,
    int inputWidth,
    int inputChannel,
    int kernelNumber,
    int kernelHeight,
    int kernelWidth,
    int outputHeight,
    int outputWidth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Output width parallel
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Output height parallel
    int z = blockIdx.z;                            // Kernel number parallel

    if (x >= outputWidth || y >= outputHeight) {
        return;
    }

    for (int c = 0; c < inputChannel; c++) {
        for (int h = 0; h < kernelHeight; h++) {
            for (int w = 0; w < kernelWidth; w++) {
                output[z * outputHeight * outputWidth + y * outputWidth + x] +=
                    input[c * inputHeight * inputWidth + (y + h) * inputWidth + (x + w)] * kernel[z * kernelHeight * kernelWidth * inputChannel + c * kernelHeight * kernelWidth + h * kernelWidth + w];
            }
        }
    }
}

int main() {
    hipError_t cudaStatus;
    printf("Input shape: (%d, %d, %d)\n", INPUT_CHANNEL, INPUT_HEIGHT, INPUT_WIDTH);
    printf("Kernel shape: (%d, %d, %d, %d)\n", KERNEL_NUMBER, KERNEL_CHANNEL, KERNEL_HEIGHT, KERNEL_WIDTH);
    printf("Output shape: (%d, %d, %d)\n", KERNEL_NUMBER, OUTPUT_HEIGHT, OUTPUT_WIDTH);

    hipEvent_t start_malloc, stop_malloc;
    hipEventCreate(&start_malloc);
    hipEventCreate(&stop_malloc);
    hipEventRecord(start_malloc);

    double *input = (double *)malloc(INPUT_CHANNEL * INPUT_HEIGHT * INPUT_WIDTH * sizeof(double));
    double *output = (double *)malloc(KERNEL_NUMBER * OUTPUT_HEIGHT * OUTPUT_WIDTH * sizeof(double));
    double *kernel = (double *)malloc(KERNEL_NUMBER * KERNEL_CHANNEL * KERNEL_HEIGHT * KERNEL_WIDTH * sizeof(double));

    omp_set_num_threads(omp_get_max_threads());
    printf("Number of threads: %d\n", omp_get_max_threads());

// Initialize input
#pragma omp parallel for simd
    for (int i = 0; i < INPUT_CHANNEL * INPUT_HEIGHT * INPUT_WIDTH; i++) {
        input[i] = i / 100013.0;
    }

// Initialize kernel
#pragma omp parallel for simd
    for (int i = 0; i < KERNEL_NUMBER * KERNEL_CHANNEL * KERNEL_HEIGHT * KERNEL_WIDTH; i++) {
        kernel[i] = i / 100013.0;
    }

// Initialize output
#pragma omp parallel for simd
    for (int i = 0; i < KERNEL_NUMBER * OUTPUT_HEIGHT * OUTPUT_WIDTH; i++) {
        output[i] = 0;
    }

    hipEventRecord(stop_malloc);
    hipEventSynchronize(stop_malloc);
    float malloc_time = 0;
    hipEventElapsedTime(&malloc_time, start_malloc, stop_malloc);
    printf("Malloc time: %.3f ms\n", malloc_time);
    hipEventDestroy(start_malloc);
    hipEventDestroy(stop_malloc);

    // Load to device
    double *d_input, *d_output, *d_kernel;
    cudaStatus = hipMalloc(&d_input, INPUT_CHANNEL * INPUT_HEIGHT * INPUT_WIDTH * sizeof(double));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc input failed\n");
        return 1;
    }
    cudaStatus = hipMalloc(&d_output, KERNEL_NUMBER * OUTPUT_HEIGHT * OUTPUT_WIDTH * sizeof(double));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc output failed\n");
        return 1;
    }
    cudaStatus = hipMalloc(&d_kernel, KERNEL_NUMBER * KERNEL_CHANNEL * KERNEL_HEIGHT * KERNEL_WIDTH * sizeof(double));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc kernel failed\n");
        return 1;
    }

    cudaStatus = hipMemcpy(d_input, input, INPUT_CHANNEL * INPUT_HEIGHT * INPUT_WIDTH * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy input failed\n");
        return 1;
    }
    cudaStatus = hipMemcpy(d_kernel, kernel, KERNEL_NUMBER * KERNEL_CHANNEL * KERNEL_HEIGHT * KERNEL_WIDTH * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy kernel failed\n");
        return 1;
    }

    dim3 block(16, 16, 1);                                                                                     // xyz
    dim3 grid((OUTPUT_WIDTH + block.x - 1) / block.x, (OUTPUT_HEIGHT + block.y - 1) / block.y, KERNEL_NUMBER); // grid

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    convolution<<<grid, block>>>(d_input, d_output, d_kernel, INPUT_HEIGHT, INPUT_WIDTH, INPUT_CHANNEL, KERNEL_NUMBER, KERNEL_HEIGHT, KERNEL_WIDTH, OUTPUT_HEIGHT, OUTPUT_WIDTH);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %.3f ms\n", milliseconds);
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    printf("Memory: Used %.2f MB, Free %.2f MB, Total %.2f MB\n",
           (total_mem - free_mem) / 1024.0 / 1024.0,
           free_mem / 1024.0 / 1024.0,
           total_mem / 1024.0 / 1024.0);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy back to host
    hipMemcpy(output, d_output, KERNEL_NUMBER * OUTPUT_HEIGHT * OUTPUT_WIDTH * sizeof(double), hipMemcpyDeviceToHost);

    // Output to file
    const char *filename = "output.bit";
    FILE *fp = fopen(filename, "wb");
    if (!fp) {
        printf("Can't open file %s\n", filename);
        return 1;
    }
    fwrite(output, sizeof(double), KERNEL_NUMBER * OUTPUT_HEIGHT * OUTPUT_WIDTH, fp);
    fclose(fp);

    free(input);
    free(output);
    free(kernel);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);

    return 0;
}